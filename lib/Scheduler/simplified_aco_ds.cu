#include "hip/hip_runtime.h"
#include "opt-sched/Scheduler/logger.h"
#include "opt-sched/Scheduler/simplified_aco_ds.h"
#include "opt-sched/Scheduler/register.h"
#include "opt-sched/Scheduler/data_dep.h"
#include "opt-sched/Scheduler/sched_basic_data.h"
#include "opt-sched/Scheduler/machine_model.h"
#include <algorithm>
#include <cstddef>
#include <utility>
//aco simplified ds impl

using namespace llvm::opt_sched;

//use the log message macro to make GPU porting easier
#define LOG_MESSAGE(...) Logger::Info(__VA_ARGS__)

// ----
// ACOReadyList
// ----

ACOReadyList::ACOReadyList() {
  InstrCount = 0;
  CurrentSize = 0;
  CurrentCapacity = PrimaryBufferCapacity = 0;
  Overflowed = false;

  // create new allocations for the data
  IntAllocation = nullptr;
  HeurAllocation = nullptr;
  ScoreAllocation = nullptr;

  //build shortcut pointers
  InstrBase = nullptr;
  ReadyOnBase = nullptr;
  HeurBase = nullptr;
  ScoreBase = nullptr;

}

ACOReadyList::ACOReadyList(InstCount RegionSize) {
  InstrCount = RegionSize;
  CurrentSize = 0;
  CurrentCapacity = PrimaryBufferCapacity = computePrimaryCapacity(InstrCount);
  Overflowed = false;

  // create new allocations for the data
  IntAllocation = new InstCount[2*CurrentCapacity];
  HeurAllocation = new HeurType[CurrentCapacity];
  ScoreAllocation = new pheromone_t[CurrentCapacity];

  //build shortcut pointers
  InstrBase = IntAllocation;
  ReadyOnBase = IntAllocation + CurrentCapacity;
  HeurBase = HeurAllocation;
  ScoreBase = ScoreAllocation;
}

ACOReadyList::ACOReadyList(const ACOReadyList &Other) {
  InstrCount = Other.InstrCount;
  PrimaryBufferCapacity = Other.PrimaryBufferCapacity;
  Overflowed = Other.Overflowed;
  CurrentCapacity = Other.CurrentCapacity;
  CurrentSize = Other.CurrentSize;

  // create new allocations for the data
  IntAllocation = new InstCount[2*CurrentCapacity];
  HeurAllocation = new HeurType[CurrentCapacity];
  ScoreAllocation = new pheromone_t[CurrentCapacity];

  //build shortcut pointers
  InstrBase = IntAllocation;
  ReadyOnBase = IntAllocation + CurrentCapacity;
  HeurBase = HeurAllocation;
  ScoreBase = ScoreAllocation;

  // copy the allocation's entries
  for (InstCount I = 0; I < CurrentSize; ++I) {
    InstrBase[I] = Other.InstrBase[I];
    ReadyOnBase[I] = Other.ReadyOnBase[I];
    HeurBase[I] = Other.HeurBase[I];
    ScoreBase[I] = Other.ScoreBase[I];
  }
}

ACOReadyList &ACOReadyList::operator=(const ACOReadyList &Other) {
  InstrCount = Other.InstrCount;
  PrimaryBufferCapacity = Other.PrimaryBufferCapacity;
  Overflowed = Other.Overflowed;
  CurrentCapacity = Other.CurrentCapacity;
  CurrentSize = Other.CurrentSize;

  // delete current allocations
  delete[] IntAllocation;
  delete[] HeurAllocation;
  delete[] ScoreAllocation;

  // create new allocations for the data
  IntAllocation = new InstCount[2*CurrentCapacity];
  HeurAllocation = new HeurType[CurrentCapacity];
  ScoreAllocation = new pheromone_t[CurrentCapacity];

  //build shortcut pointers
  InstrBase = IntAllocation;
  ReadyOnBase = IntAllocation + CurrentCapacity;
  HeurBase = HeurAllocation;
  ScoreBase = ScoreAllocation;

  // copy over the allocation's entries
  for (InstCount I = 0; I < CurrentSize; ++I) {
    InstrBase[I] = Other.InstrBase[I];
    ReadyOnBase[I] = Other.ReadyOnBase[I];
    HeurBase[I] = Other.HeurBase[I];
    ScoreBase[I] = Other.ScoreBase[I];
  }

  return *this;
}

ACOReadyList::ACOReadyList(ACOReadyList &&Other) noexcept {
  InstrCount = Other.InstrCount;
  PrimaryBufferCapacity = Other.PrimaryBufferCapacity;
  Overflowed = Other.Overflowed;
  CurrentCapacity = Other.CurrentCapacity;
  CurrentSize = Other.CurrentSize;

  // copy over the old ready lists allocations and set them to NULL
  // so that the data we took won't get deleted
  IntAllocation = Other.IntAllocation;
  HeurAllocation = Other.HeurAllocation;
  ScoreAllocation = Other.ScoreAllocation;
  Other.IntAllocation = nullptr;
  Other.HeurAllocation = nullptr;
  Other.ScoreAllocation = nullptr;

  InstrBase = Other.InstrBase;
  ReadyOnBase = Other.ReadyOnBase;
  HeurBase = Other.HeurBase;
  ScoreBase = Other.ScoreBase;
}

ACOReadyList &ACOReadyList::operator=(ACOReadyList &&Other) noexcept {
  InstrCount = Other.InstrCount;
  PrimaryBufferCapacity = Other.PrimaryBufferCapacity;
  Overflowed = Other.Overflowed;
  CurrentCapacity = Other.CurrentCapacity;
  CurrentSize = Other.CurrentSize;

  // swap the allocations to give Other our allocations to delete
  std::swap(IntAllocation, Other.IntAllocation);
  std::swap(HeurAllocation, Other.HeurAllocation);
  std::swap(ScoreAllocation, Other.ScoreAllocation);

  InstrBase = Other.InstrBase;
  ReadyOnBase = Other.ReadyOnBase;
  HeurBase = Other.HeurBase;
  ScoreBase = Other.ScoreBase;

  return *this;
}

ACOReadyList::~ACOReadyList() {
  delete[] IntAllocation;
  delete[] HeurAllocation;
  delete[] ScoreAllocation;
}


// This is just a heuristic for the ready list size.
// A better function should be chosen experimentally
InstCount ACOReadyList::computePrimaryCapacity(InstCount RegionSize) {
  //return std::max(32, RegionSize/4);
  return RegionSize;
}

__host__ __device__
void ACOReadyList::addInstructionToReadyList(const ACOReadyListEntry &Entry) {
  #ifdef __CUDA_ARCH__
    if (CurrentSize == CurrentCapacity) {
      printf("Ready List ran out of capacity and needs to be resized");
      exit(1);
    } else {
      //add the instruction to the ready list
      InstrBase[CurrentSize*numThreads_ + GLOBALTID] = Entry.InstId;
      ReadyOnBase[CurrentSize*numThreads_ + GLOBALTID] = Entry.ReadyOn;
      HeurBase[CurrentSize*numThreads_ + GLOBALTID] = Entry.Heuristic;
      ScoreBase[CurrentSize*numThreads_ + GLOBALTID] = Entry.Score;
      ++CurrentSize;
    }

    /*if (CurrentSize == CurrentCapacity) {
      int OldCap = CurrentCapacity;
      bool PrevOverflowed = Overflowed;

      // get a new allocation to put the data in
      // The expansion formula is to make the new allocation 1.5 times the size of the old one
      // consider making this formula more aggressive
      int NewCap = (OldCap + OldCap/2 + 1) * numThreads;
      InstCount *NewIntFallback = new InstCount[2*NewCap];
      HeurType *NewHeurFallback = new HeurType[NewCap];
      pheromone_t *NewScoreFallback = new pheromone_t[NewCap];

      // copy the data
      InstCount NewInstrOffset = 0, NewReadyOnOffset = NewCap, HeurOffset = 0, ScoreOffset = 0;
      for (int I = 0; I < CurrentSize; ++I) {
        NewIntFallback[numThreads*(NewInstrOffset + I) + GLOBALTID] = InstrBase[numThreads*I + GLOBALTID];
        NewIntFallback[numThreads*(NewReadyOnOffset + I) + GLOBALTID] = ReadyOnBase[numThreads*I + GLOBALTID];
        NewHeurFallback[numThreads*(HeurOffset + I) + GLOBALTID] = HeurBase[numThreads*I + GLOBALTID];
        NewScoreFallback[numThreads*(ScoreOffset + I) + GLOBALTID] = ScoreBase[numThreads*I + GLOBALTID];
    }*/
  #else
    // check to see if we need to expand the allocation/get a new allocation
    if (CurrentSize == CurrentCapacity) {
      int OldCap = CurrentCapacity;
      bool PrevOverflowed = Overflowed;

      // get a new allocation to put the data in
      // The expansion formula is to make the new allocation 1.5 times the size of the old one
      // consider making this formula more aggressive
      int NewCap = OldCap + OldCap/2 + 1;
      InstCount *NewIntFallback = new InstCount[2*NewCap];
      HeurType *NewHeurFallback = new HeurType[NewCap];
      pheromone_t *NewScoreFallback = new pheromone_t[NewCap];

      // copy the data
      InstCount NewInstrOffset = 0, NewReadyOnOffset = NewCap, HeurOffset = 0, ScoreOffset = 0;
      for (int I = 0; I < CurrentSize; ++I) {
        NewIntFallback[NewInstrOffset + I] = InstrBase[I];
        NewIntFallback[NewReadyOnOffset + I] = ReadyOnBase[I];
        NewHeurFallback[HeurOffset + I] = HeurBase[I];
        NewScoreFallback[ScoreOffset + I] = ScoreBase[I];
      }

      //delete the old allocations
      delete[] IntAllocation;
      delete[] HeurAllocation;
      delete[] ScoreAllocation;

      //copy the new allocations
      IntAllocation = NewIntFallback;
      HeurAllocation = NewHeurFallback;
      ScoreAllocation = NewScoreFallback;

      // update/recompute pointers and other values
      InstrBase = IntAllocation + NewInstrOffset;
      ReadyOnBase = IntAllocation + NewReadyOnOffset;
      HeurBase = HeurAllocation + HeurOffset;
      ScoreBase = ScoreAllocation + ScoreOffset;
      Overflowed = true;
      CurrentCapacity = NewCap;

      //print out a notice/error message
      //Welp this may be a performance disaster if this is happening too much
      LOG_MESSAGE("Overflowed ReadyList capacity. Old Cap:%d, New Cap:%d, Primary Cap:%d, Prev Overflowed:%B", OldCap, NewCap, PrimaryBufferCapacity, PrevOverflowed);
    }

    //add the instruction to the ready list
    InstrBase[CurrentSize] = Entry.InstId;
    ReadyOnBase[CurrentSize] = Entry.ReadyOn;
    HeurBase[CurrentSize] = Entry.Heuristic;
    ScoreBase[CurrentSize] = Entry.Score;
    ++CurrentSize;
  #endif
}

// We copy the instruction at the end of the array to the instruction at the target index
// then we decrement the Ready List's CurrentSize
// This function has undefined behavior if CurrentSize == 0
__host__ __device__
ACOReadyListEntry ACOReadyList::removeInstructionAtIndex(InstCount Indx) {
  assert(CurrentSize <= 0 || Indx >= CurrentSize || Indx < 0);
  #ifdef __CUDA_ARCH__
    ACOReadyListEntry E{InstrBase[Indx*numThreads_ + GLOBALTID], 
                        ReadyOnBase[Indx*numThreads_ + GLOBALTID], 
                        HeurBase[Indx*numThreads_ + GLOBALTID], 
                        ScoreBase[Indx*numThreads_ + GLOBALTID]};
    InstCount EndIndx = --CurrentSize;
    InstrBase[Indx*numThreads_ + GLOBALTID] = InstrBase[EndIndx*numThreads_ + GLOBALTID];
    ReadyOnBase[Indx*numThreads_ + GLOBALTID] = ReadyOnBase[EndIndx*numThreads_ + GLOBALTID];
    HeurBase[Indx*numThreads_ + GLOBALTID] = HeurBase[EndIndx*numThreads_ + GLOBALTID];
    ScoreBase[Indx*numThreads_ + GLOBALTID] = ScoreBase[EndIndx*numThreads_ + GLOBALTID];
    return E;
  #else
    ACOReadyListEntry E{InstrBase[Indx], ReadyOnBase[Indx], HeurBase[Indx], ScoreBase[Indx]};
    InstCount EndIndx = --CurrentSize;
    InstrBase[Indx] = InstrBase[EndIndx];
    ReadyOnBase[Indx] = ReadyOnBase[EndIndx];
    HeurBase[Indx] = HeurBase[EndIndx];
    ScoreBase[Indx] = ScoreBase[EndIndx];
    return E;
  #endif
}

void ACOReadyList::AllocDevArraysForParallelACO(int numThreads) {
  size_t memSize;
  numThreads_ = numThreads;

  // Alloc dev array for dev_IntAllocation
  memSize = sizeof(InstCount*) * CurrentCapacity * numThreads_ * 2;
  gpuErrchk(hipMallocManaged(&dev_IntAllocation, memSize));

  // Alloc dev array for dev_HeurAllocation
  memSize = sizeof(HeurType*) * CurrentCapacity * numThreads_;
  gpuErrchk(hipMallocManaged(&dev_HeurAllocation, memSize));

  // Alloc dev array for dev_ScoreAllocation
  memSize = sizeof(pheromone_t*) * CurrentCapacity * numThreads_;
  gpuErrchk(hipMallocManaged(&dev_ScoreAllocation, memSize));

  //build shortcut pointers
  InstrBase = dev_IntAllocation;
  ReadyOnBase = dev_IntAllocation + CurrentCapacity*numThreads;
  HeurBase = dev_HeurAllocation;
  ScoreBase = dev_ScoreAllocation;

  // prefetch memory used with hipMallocManaged
  memSize = sizeof(InstCount*) * numThreads_ * 2;
  gpuErrchk(hipMemPrefetchAsync(dev_IntAllocation, memSize, 0));

  memSize = sizeof(HeurType*) * numThreads_;
  gpuErrchk(hipMemPrefetchAsync(dev_HeurAllocation, memSize, 0));

  memSize = sizeof(pheromone_t*) * numThreads_;
  gpuErrchk(hipMemPrefetchAsync(dev_ScoreAllocation, memSize, 0));

  /*// Alloc dev array for dev_IntAllocation
  memSize = sizeof(InstCount*) * numThreads_ * 2;
  gpuErrchk(hipMallocManaged(&dev_IntAllocation, memSize));

  // Alloc dev array for dev_HeurAllocation
  memSize = sizeof(HeurType*) * numThreads_;
  gpuErrchk(hipMallocManaged(&dev_HeurAllocation, memSize));

  // Alloc dev array for dev_ScoreAllocation
  memSize = sizeof(pheromone_t*) * numThreads_;
  gpuErrchk(hipMallocManaged(&dev_ScoreAllocation, memSize)); */
}

void ACOReadyList::CopyPointersToDevice(ACOReadyList *dev_acoRdyLst, int numThreads) {
  size_t memSize;

  // copy over arrays
  memSize = sizeof(InstCount*) * numThreads * 2;
  for (int i = 0; i < numThreads; i++) {
    gpuErrchk(hipMemcpy(&dev_acoRdyLst->dev_IntAllocation[i], IntAllocation, memSize,
	  	         hipMemcpyHostToDevice));
  }
  memSize = sizeof(HeurType*) * numThreads;
  for (int i = 0; i < numThreads; i++) {
    gpuErrchk(hipMemcpy(&dev_acoRdyLst->dev_HeurAllocation[i], HeurAllocation, memSize,
	  	         hipMemcpyHostToDevice));
  }
  memSize = sizeof(pheromone_t*) * numThreads;
  for (int i = 0; i < numThreads; i++) {
    gpuErrchk(hipMemcpy(&dev_acoRdyLst->dev_ScoreAllocation[i], ScoreAllocation, memSize,
	  	         hipMemcpyHostToDevice));
  }

  // Alloc elmnts for each array
  InstCount *temp_intArr;
  memSize = sizeof(InstCount) * CurrentCapacity * numThreads * 2;
  gpuErrchk(hipMalloc(&temp_intArr, memSize));

  HeurType *temp_HeurArr;
  memSize = sizeof(HeurType) * CurrentCapacity * numThreads;
  gpuErrchk(hipMalloc(&temp_HeurArr, memSize));

  pheromone_t *temp_scoreArr;
  memSize = sizeof(pheromone_t) * CurrentCapacity * numThreads;
  gpuErrchk(hipMalloc(&temp_scoreArr, memSize));

  // assign a chunk of each large array to each array
  for (int i = 0; i < numThreads; i++) {
    dev_acoRdyLst->dev_IntAllocation[i] = temp_intArr[i*CurrentCapacity];
    dev_acoRdyLst->dev_IntAllocation[i + numThreads] = temp_intArr[i*CurrentCapacity + CurrentCapacity*numThreads];
    dev_acoRdyLst->dev_HeurAllocation[i] = temp_HeurArr[i*CurrentCapacity];
    dev_acoRdyLst->dev_ScoreAllocation[i] = temp_scoreArr[i*CurrentCapacity];
  }

  //build shortcut pointers
  InstrBase = dev_IntAllocation;
  ReadyOnBase = dev_IntAllocation + CurrentCapacity*numThreads;
  HeurBase = dev_HeurAllocation;
  ScoreBase = dev_ScoreAllocation;

  // prefetch memory used with hipMallocManaged
  memSize = sizeof(InstCount*) * numThreads_ * 2;
  gpuErrchk(hipMemPrefetchAsync(dev_IntAllocation, memSize, 0));

  memSize = sizeof(HeurType*) * numThreads_;
  gpuErrchk(hipMemPrefetchAsync(dev_HeurAllocation, memSize, 0));

  memSize = sizeof(pheromone_t*) * numThreads_;
  gpuErrchk(hipMemPrefetchAsync(dev_ScoreAllocation, memSize, 0));
}

void ACOReadyList::FreeDevicePointers() {
  hipFree(dev_IntAllocation);
  hipFree(dev_HeurAllocation);
  hipFree(dev_ScoreAllocation);
}